//===----------------------------------------------------------------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//

// <math.h>

// This file was copied from libc++'s test suite, then modified to test CUDA.
// For the most part, this consists of adding __device__ attributes and
// deleting long double.

// This test requires C++11 (it's mostly decltype checks).
#if __cplusplus >= 201103L


#include <hip/hip_runtime.h>
#include <math.h>
#include <type_traits>
#include <cassert>
#include <stdio.h>

// See PR21083
// Ambiguous is a user-defined type that defines its own overloads of cmath
// functions. When the std overloads are candidates too (by using or adl),
// they should not interfere.
struct Ambiguous : std::true_type { // ADL
    __device__ operator float () { return 0.f; }
    __device__ operator double () { return 0.; }
};
__device__ Ambiguous abs(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous acos(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous asin(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous atan(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous atan2(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous ceil(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous cos(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous cosh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous exp(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fabs(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous floor(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fmod(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous frexp(Ambiguous, int*){ return Ambiguous(); }
__device__ Ambiguous ldexp(Ambiguous, int){ return Ambiguous(); }
__device__ Ambiguous log(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous log10(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous modf(Ambiguous, Ambiguous*){ return Ambiguous(); }
__device__ Ambiguous pow(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous sin(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous sinh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous sqrt(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous tan(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous tanh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous signbit(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fpclassify(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isfinite(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isnormal(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isgreater(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isgreaterequal(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isless(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous islessequal(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous islessgreater(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous isunordered(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous acosh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous asinh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous atanh(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous cbrt(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous copysign(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous erf(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous erfc(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous exp2(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous expm1(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fdim(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fma(Ambiguous, Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fmax(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous fmin(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous hypot(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous ilogb(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous lgamma(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous llrint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous llround(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous log1p(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous log2(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous logb(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous lrint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous lround(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous nearbyint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous nextafter(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous remainder(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous remquo(Ambiguous, Ambiguous, int*){ return Ambiguous(); }
__device__ Ambiguous rint(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous round(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous scalbln(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous scalbn(Ambiguous, Ambiguous){ return Ambiguous(); }
__device__ Ambiguous tgamma(Ambiguous){ return Ambiguous(); }
__device__ Ambiguous trunc(Ambiguous){ return Ambiguous(); }


// helper function to prevent compiler constant-folding test inputs.

template <typename T>
__device__ T V(T input) {
  volatile T tmp = input;
  return tmp;
}

__device__ void test_abs()
{
    static_assert((std::is_same<decltype(abs((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(abs((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(abs(Ambiguous())), Ambiguous>::value), "");
    assert(abs(V(-1)) == 1);
    assert(abs(V(-1.)) == 1);
    assert(abs(V(-1.f)) == 1);
}

__device__ void test_acos()
{
    static_assert((std::is_same<decltype(acos((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(acos((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acos((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosf(0)), float>::value), "");
    static_assert((std::is_same<decltype(acos(Ambiguous())), Ambiguous>::value), "");
    assert(acos(V(1)) == 0);
    assert(acos(V(1.)) == 0);
    assert(acos(V(1.f)) == 0);
}

__device__ void test_asin()
{
    static_assert((std::is_same<decltype(asin((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(asin((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asin((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinf(0)), float>::value), "");
    static_assert((std::is_same<decltype(asin(Ambiguous())), Ambiguous>::value), "");
    assert(asin(V(0)) == 0);
    assert(asin(V(0.)) == 0);
    assert(asin(V(0.f)) == 0);
}

__device__ void test_atan()
{
    static_assert((std::is_same<decltype(atan((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(atan((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanf(0)), float>::value), "");
    static_assert((std::is_same<decltype(atan(Ambiguous())), Ambiguous>::value), "");
    assert(atan(V(0)) == 0);
    assert(atan(V(0.)) == 0);
    assert(atan(V(0.f)) == 0);
}

__device__ void test_atan2()
{
    static_assert((std::is_same<decltype(atan2((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(atan2((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2f(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(atan2((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atan2(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(atan2(V(0), 1) == 0);
    assert(atan2(V(0), 1.) == 0);
    assert(atan2(V(0), 1.f) == 0);

    assert(atan2(V(0.), 1) == 0);
    assert(atan2(V(0.), 1.) == 0);
    assert(atan2(V(0.), 1.f) == 0);

    assert(atan2(V(0.f), 1) == 0);
    assert(atan2(V(0.f), 1.) == 0);
    assert(atan2(V(0.f), 1.f) == 0);
}

__device__ void test_ceil()
{
    static_assert((std::is_same<decltype(ceil((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(ceil((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceil((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(ceilf(0)), float>::value), "");
    static_assert((std::is_same<decltype(ceil(Ambiguous())), Ambiguous>::value), "");
    assert(ceil(V(0)) == 0);
    assert(ceil(V(0.)) == 0);
    assert(ceil(V(0.f)) == 0);
}

__device__ void test_cos()
{
    static_assert((std::is_same<decltype(cos((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(cos((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cos((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosf(0)), float>::value), "");
    static_assert((std::is_same<decltype(cos(Ambiguous())), Ambiguous>::value), "");
    assert(cos(V(0)) == 1);
    assert(cos(V(0.)) == 1);
    assert(cos(V(0.f)) == 1);
}

__device__ void test_cosh()
{
    static_assert((std::is_same<decltype(cosh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(cosh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cosh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(coshf(0)), float>::value), "");
    static_assert((std::is_same<decltype(cosh(Ambiguous())), Ambiguous>::value), "");
    assert(cosh(V(0)) == 1);
    assert(cosh(V(0.)) == 1);
    assert(cosh(V(0.f)) == 1);
}

__device__ void test_exp()
{
    static_assert((std::is_same<decltype(exp((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(exp((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(expf(0)), float>::value), "");
    static_assert((std::is_same<decltype(exp(Ambiguous())), Ambiguous>::value), "");
    assert(exp(V(0)) == 1);
    assert(exp(V(0.)) == 1);
    assert(exp(V(0.f)) == 1);
}

__device__ void test_fabs()
{
    static_assert((std::is_same<decltype(fabs((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(fabs((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabs((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fabsf(0.0f)), float>::value), "");
    static_assert((std::is_same<decltype(fabs(Ambiguous())), Ambiguous>::value), "");
    assert(fabs(V(-1)) == 1);
    assert(fabs(V(-1.)) == 1);
    assert(fabs(V(-1.f)) == 1);
}

__device__ void test_floor()
{
    static_assert((std::is_same<decltype(floor((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(floor((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(floor((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(floorf(0)), float>::value), "");
    static_assert((std::is_same<decltype(floor(Ambiguous())), Ambiguous>::value), "");
    assert(floor(V(1)) == 1);
    assert(floor(V(1.)) == 1);
    assert(floor(V(1.f)) == 1);
}

__device__ void test_fmod()
{
    static_assert((std::is_same<decltype(fmod((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(fmod((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmodf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(fmod((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmod(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(fmod(V(1.5), 1) == .5);
    assert(fmod(V(1.5), 1.) == .5);
    assert(fmod(V(1.5), 1.f) == .5);

    assert(fmod(V(1.5f), 1) == .5);
    assert(fmod(V(1.5f), 1.) == .5);
    assert(fmod(V(1.5f), 1.f) == .5);

    assert(fmod(V(2), 1) == 0);
    assert(fmod(V(2), 1.) == 0);
    assert(fmod(V(2), 1.f) == 0);
}

__device__ void test_frexp()
{
    int ip;
    static_assert((std::is_same<decltype(frexp((float)0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(frexp((bool)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((unsigned short)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((unsigned int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((unsigned long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((unsigned long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexp((double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(frexpf(0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(frexp(Ambiguous(), &ip)), Ambiguous>::value), "");
    assert(frexp(V(0), &ip) == 0);
    assert(frexp(V(0.), &ip) == 0);
    assert(frexp(V(0.f), &ip) == 0);
}

__device__ void test_ldexp()
{
    int ip = 1;
    static_assert((std::is_same<decltype(ldexp((float)0, ip)), float>::value), "");
    static_assert((std::is_same<decltype(ldexp((bool)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((unsigned short)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((int)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((unsigned int)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((unsigned long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((long long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((unsigned long long)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexp((double)0, ip)), double>::value), "");
    static_assert((std::is_same<decltype(ldexpf(0, ip)), float>::value), "");
    static_assert((std::is_same<decltype(ldexp(Ambiguous(), ip)), Ambiguous>::value), "");
    assert(ldexp(V(1), ip) == 2);
    assert(ldexp(V(1.), ip) == 2);
    assert(ldexp(V(1.f), ip) == 2);
}

__device__ void test_log()
{
    static_assert((std::is_same<decltype(log((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(log((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(logf(0)), float>::value), "");
    static_assert((std::is_same<decltype(log(Ambiguous())), Ambiguous>::value), "");
    assert(log(V(1)) == 0);
    assert(log(V(1.)) == 0);
    assert(log(V(1.f)) == 0);
}

__device__ void test_log10()
{
    static_assert((std::is_same<decltype(log10((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(log10((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(log10f(0)), float>::value), "");
    static_assert((std::is_same<decltype(log10(Ambiguous())), Ambiguous>::value), "");
    assert(log10(V(1)) == 0);
    assert(log10(V(1.)) == 0);
    assert(log10(V(1.f)) == 0);
}

__device__ void test_modf()
{
    static_assert((std::is_same<decltype(modf((float)0, (float*)0)), float>::value), "");
    static_assert((std::is_same<decltype(modf((double)0, (double*)0)), double>::value), "");
    static_assert((std::is_same<decltype(modff(0, (float*)0)), float>::value), "");
    static_assert((std::is_same<decltype(modf(Ambiguous(), (Ambiguous*)0)), Ambiguous>::value), "");
    double i;
    assert(modf(V(1), &i) == 0);
    assert(modf(V(1.), &i) == 0);
    assert(modf(V(1.f), &i) == 0);
}

__device__ void test_pow()
{
    static_assert((std::is_same<decltype(pow((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(pow((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(powf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(pow((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(pow(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(pow(V(1), 1) == 1);
    assert(pow(V(1.), 1) == 1);
    assert(pow(V(1.f), 1) == 1);

    assert(pow(V(1), 1.) == 1);
    assert(pow(V(1.), 1.) == 1);
    assert(pow(V(1.f), 1.) == 1);

    assert(pow(V(1), 1.f) == 1);
    assert(pow(V(1.), 1.f) == 1);
    assert(pow(V(1.f), 1.f) == 1);
}

__device__ void test_sin()
{
    static_assert((std::is_same<decltype(sin((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(sin((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sin((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinf(0)), float>::value), "");
    static_assert((std::is_same<decltype(sin(Ambiguous())), Ambiguous>::value), "");
    assert(sin(0) == 0);
    assert(sin(0.) == 0);
    assert(sin(0.f) == 0);
}

__device__ void test_sinh()
{
    static_assert((std::is_same<decltype(sinh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(sinh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(sinhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(sinh(Ambiguous())), Ambiguous>::value), "");
    assert(sinh(V(0)) == 0);
    assert(sinh(V(0.)) == 0);
    assert(sinh(V(0.f)) == 0);
}

__device__ void test_sqrt()
{
    static_assert((std::is_same<decltype(sqrt((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(sqrt((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrt((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(sqrtf(0)), float>::value), "");
    static_assert((std::is_same<decltype(sqrt(Ambiguous())), Ambiguous>::value), "");
    assert(sqrt(V(4)) == 2);
    assert(sqrt(V(4.)) == 2);
    assert(sqrt(V(4.f)) == 2);
}

__device__ void test_tan()
{
    static_assert((std::is_same<decltype(tan((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(tan((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tan((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanf(0)), float>::value), "");
    static_assert((std::is_same<decltype(tan(Ambiguous())), Ambiguous>::value), "");
    assert(tan(V(0)) == 0);
    assert(tan(V(0.)) == 0);
    assert(tan(V(0.f)) == 0);
}

__device__ void test_tanh()
{
    static_assert((std::is_same<decltype(tanh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(tanh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(tanhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(tanh(Ambiguous())), Ambiguous>::value), "");
    assert(tanh(V(0)) == 0);
    assert(tanh(V(0.)) == 0);
    assert(tanh(V(0.f)) == 0);
}

__device__ void test_signbit()
{
#ifdef signbit
#error signbit defined
#endif
    static_assert((std::is_same<decltype(signbit((float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(signbit((double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(signbit(0)), bool>::value), "");
    static_assert((std::is_same<decltype(signbit(Ambiguous())), Ambiguous>::value), "");
    assert(signbit(V(-1)) == true);
    assert(signbit(V(-1.)) == true);
    assert(signbit(V(-1.f)) == true);
}

__device__ void test_fpclassify()
{
#ifdef fpclassify
#error fpclassify defined
#endif
    static_assert((std::is_same<decltype(fpclassify((float)0)), int>::value), "");
    static_assert((std::is_same<decltype(fpclassify((double)0)), int>::value), "");
    static_assert((std::is_same<decltype(fpclassify(0)), int>::value), "");
    static_assert((std::is_same<decltype(fpclassify(Ambiguous())), Ambiguous>::value), "");
    assert(fpclassify(V(-1)) == FP_NORMAL);
    assert(fpclassify(V(-1.)) == FP_NORMAL);
    assert(fpclassify(V(-1.f)) == FP_NORMAL);
}

__device__ void test_isfinite()
{
#ifdef isfinite
#error isfinite defined
#endif
    static_assert((std::is_same<decltype(isfinite((float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isfinite((double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isfinite(0)), bool>::value), "");
    static_assert((std::is_same<decltype(isfinite(Ambiguous())), Ambiguous>::value), "");
    assert(isfinite(V(-1)) == true);
    assert(isfinite(V(-1.)) == true);
    assert(isfinite(V(-1.f)) == true);
}

__device__ void test_isnormal()
{
#ifdef isnormal
#error isnormal defined
#endif
    static_assert((std::is_same<decltype(isnormal((float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isnormal((double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isnormal(0)), bool>::value), "");
    static_assert((std::is_same<decltype(isnormal(Ambiguous())), Ambiguous>::value), "");
    assert(std::isnormal(V(-1)) == true);
    assert(std::isnormal(V(-1.)) == true);
    assert(std::isnormal(V(-1.f)) == true);
}

__device__ void test_isgreater()
{
#ifdef isgreater
#error isgreater defined
#endif
    static_assert((std::is_same<decltype(isgreater((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreater((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreater((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreater((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreater(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreater(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isgreater(V(-1), 0) == false);
    assert(std::isgreater(V(-1), 0.) == false);
    assert(std::isgreater(V(-1), 0.f) == false);

    assert(std::isgreater(V(-1.), 0) == false);
    assert(std::isgreater(V(-1.), 0.) == false);
    assert(std::isgreater(V(-1.), 0.f) == false);

    assert(std::isgreater(V(-1.f), 0) == false);
    assert(std::isgreater(V(-1.f), 0.) == false);
    assert(std::isgreater(V(-1.f), 0.f) == false);
}

__device__ void test_isgreaterequal()
{
#ifdef isgreaterequal
#error isgreaterequal defined
#endif
    static_assert((std::is_same<decltype(isgreaterequal((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreaterequal((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreaterequal((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreaterequal((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreaterequal(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isgreaterequal(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isgreaterequal(V(-1), 0) == false);
    assert(std::isgreaterequal(V(-1), 0.) == false);
    assert(std::isgreaterequal(V(-1), 0.f) == false);

    assert(std::isgreaterequal(V(-1.), 0) == false);
    assert(std::isgreaterequal(V(-1.), 0.) == false);
    assert(std::isgreaterequal(V(-1.), 0.f) == false);

    assert(std::isgreaterequal(V(-1.f), 0) == false);
    assert(std::isgreaterequal(V(-1.f), 0.) == false);
    assert(std::isgreaterequal(V(-1.f), 0.f) == false);
}

__device__ void test_isinf()
{
#ifdef isinf
#error isinf defined
#endif
    static_assert((std::is_same<decltype(isinf((float)0)), bool>::value), "");

    typedef decltype(isinf((double)0)) DoubleRetType;
#ifndef __linux__
    static_assert((std::is_same<DoubleRetType, bool>::value), "");
#else
    // GLIBC < 2.26 defines 'isinf(double)' with a return type of 'int' in
    // all C++ dialects. The test should tolerate this.
    // See: https://sourceware.org/bugzilla/show_bug.cgi?id=19439
    static_assert((std::is_same<DoubleRetType, bool>::value
                || std::is_same<DoubleRetType, int>::value), "");
#endif

    static_assert((std::is_same<decltype(isinf(0)), bool>::value), "");
    assert(std::isinf(V(-1)) == false);
    assert(std::isinf(V(-1.)) == false);
    assert(std::isinf(V(-1.f)) == false);
}

__device__ void test_isless()
{
#ifdef isless
#error isless defined
#endif
    static_assert((std::is_same<decltype(isless((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isless((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isless((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isless((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isless(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isless(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isless(V(-1), 0) == true);
    assert(std::isless(V(-1), 0.) == true);
    assert(std::isless(V(-1), 0.f) == true);

    assert(std::isless(V(-1.), 0) == true);
    assert(std::isless(V(-1.), 0.) == true);
    assert(std::isless(V(-1.), 0.f) == true);

    assert(std::isless(V(-1.f), 0) == true);
    assert(std::isless(V(-1.f), 0.) == true);
    assert(std::isless(V(-1.f), 0.f) == true);
}

__device__ void test_islessequal()
{
#ifdef islessequal
#error islessequal defined
#endif
    static_assert((std::is_same<decltype(islessequal((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessequal((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessequal((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessequal((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessequal(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessequal(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::islessequal(V(-1), 0) == true);
    assert(std::islessequal(V(-1), 0.) == true);
    assert(std::islessequal(V(-1), 0.f) == true);

    assert(std::islessequal(V(-1.), 0) == true);
    assert(std::islessequal(V(-1.), 0.) == true);
    assert(std::islessequal(V(-1.), 0.f) == true);

    assert(std::islessequal(V(-1.f), 0) == true);
    assert(std::islessequal(V(-1.f), 0.) == true);
    assert(std::islessequal(V(-1.f), 0.f) == true);
}

__device__ void test_islessgreater()
{
#ifdef islessgreater
#error islessgreater defined
#endif
    static_assert((std::is_same<decltype(islessgreater((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessgreater((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessgreater((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessgreater((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessgreater(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(islessgreater(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::islessgreater(V(-1), 0) == true);
    assert(std::islessgreater(V(-1), 0.) == true);
    assert(std::islessgreater(V(-1), 0.f) == true);

    assert(std::islessgreater(V(-1.), 0) == true);
    assert(std::islessgreater(V(-1.), 0.) == true);
    assert(std::islessgreater(V(-1.), 0.f) == true);

    assert(std::islessgreater(V(-1.f), 0) == true);
    assert(std::islessgreater(V(-1.f), 0.) == true);
    assert(std::islessgreater(V(-1.f), 0.f) == true);
}

__device__ void test_isnan()
{
#ifdef isnan
#error isnan defined
#endif
    static_assert((std::is_same<decltype(isnan((float)0)), bool>::value), "");

    typedef decltype(isnan((double)0)) DoubleRetType;
#ifndef __linux__
    static_assert((std::is_same<DoubleRetType, bool>::value), "");
#else
    // GLIBC < 2.26 defines 'isnan(double)' with a return type of 'int' in
    // all C++ dialects. The test should tolerate this.
    // See: https://sourceware.org/bugzilla/show_bug.cgi?id=19439
    static_assert((std::is_same<DoubleRetType, bool>::value
                || std::is_same<DoubleRetType, int>::value), "");
#endif

    static_assert((std::is_same<decltype(isnan(0)), bool>::value), "");
    assert(std::isnan(V(-1)) == false);
    assert(std::isnan(V(-1.)) == false);
    assert(std::isnan(V(-1.f)) == false);
}

__device__ void test_isunordered()
{
#ifdef isunordered
#error isunordered defined
#endif
    static_assert((std::is_same<decltype(isunordered((float)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isunordered((float)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isunordered((double)0, (float)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isunordered((double)0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isunordered(0, (double)0)), bool>::value), "");
    static_assert((std::is_same<decltype(isunordered(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::isunordered(V(-1), 0) == false);
    assert(std::isunordered(V(-1), 0.) == false);
    assert(std::isunordered(V(-1), 0.f) == false);

    assert(std::isunordered(V(-1.), 0) == false);
    assert(std::isunordered(V(-1.), 0.) == false);
    assert(std::isunordered(V(-1.), 0.f) == false);

    assert(std::isunordered(V(-1.f), 0) == false);
    assert(std::isunordered(V(-1.f), 0.) == false);
    assert(std::isunordered(V(-1.f), 0.f) == false);
}

__device__ void test_acosh()
{
    static_assert((std::is_same<decltype(acosh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(acosh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(acosh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(acoshf(0)), float>::value), "");
    static_assert((std::is_same<decltype(acosh(Ambiguous())), Ambiguous>::value), "");
    assert(std::acosh(V(1)) == 0);
    assert(std::acosh(V(1.)) == 0);
    assert(std::acosh(V(1.f)) == 0);
}

__device__ void test_asinh()
{
    static_assert((std::is_same<decltype(asinh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(asinh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(asinhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(asinh(Ambiguous())), Ambiguous>::value), "");
    assert(asinh(V(0)) == 0);
    assert(asinh(V(0.)) == 0);
    assert(asinh(V(0.f)) == 0);
}

__device__ void test_atanh()
{
    static_assert((std::is_same<decltype(atanh((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(atanh((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanh((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(atanhf(0)), float>::value), "");
    static_assert((std::is_same<decltype(atanh(Ambiguous())), Ambiguous>::value), "");
    assert(atanh(V(0)) == 0);
    assert(atanh(V(0.)) == 0);
    assert(atanh(V(0.f)) == 0);
}

__device__ void test_cbrt()
{
    static_assert((std::is_same<decltype(cbrt((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(cbrt((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrt((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(cbrtf(0)), float>::value), "");
    static_assert((std::is_same<decltype(cbrt(Ambiguous())), Ambiguous>::value), "");
    assert(cbrt(V(1)) == 1);
    assert(cbrt(V(1.)) == 1);
    assert(cbrt(V(1.f)) == 1);
}

__device__ void test_copysign()
{
    static_assert((std::is_same<decltype(copysign((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(copysign((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign((double)0, (double)0)), double>::value), "");

    // CUDA's copysign(float, double) currently returns a float, in violation
    // of the spec.  We can't easily change this, so accept either one.
    static_assert(
        (std::is_same<decltype(copysign((float)0, (double)0)), double>::value ||
         std::is_same<decltype(copysign((float)0, (double)0)), float>::value),
        "");

    static_assert((std::is_same<decltype(copysignf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(copysign((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(copysign(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::copysign(V(1), 1) == 1);
    assert(std::copysign(V(1.), 1) == 1);
    assert(std::copysign(V(1.f), 1) == 1);

    assert(std::copysign(V(1), 1.) == 1);
    assert(std::copysign(V(1.), 1.) == 1);
    assert(std::copysign(V(1.f), 1.) == 1);

    assert(std::copysign(V(1), 1.f) == 1);
    assert(std::copysign(V(1.), 1.f) == 1);
    assert(std::copysign(V(1.f), 1.f) == 1);
}

__device__ void test_erf()
{
    static_assert((std::is_same<decltype(erf((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(erf((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erf((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(erff(0)), float>::value), "");
    static_assert((std::is_same<decltype(erf(Ambiguous())), Ambiguous>::value), "");
    assert(erf(V(0)) == 0);
    assert(erf(V(0.)) == 0);
    assert(erf(V(0.f)) == 0);
}

__device__ void test_erfc()
{
    static_assert((std::is_same<decltype(erfc((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(erfc((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfc((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(erfcf(0)), float>::value), "");
    static_assert((std::is_same<decltype(erfc(Ambiguous())), Ambiguous>::value), "");
    assert(erfc(V(0)) == 1);
    assert(erfc(V(0.)) == 1);
    assert(erfc(V(0.f)) == 1);
}

__device__ void test_exp2()
{
    static_assert((std::is_same<decltype(exp2((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(exp2((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(exp2f(0)), float>::value), "");
    static_assert((std::is_same<decltype(exp2(Ambiguous())), Ambiguous>::value), "");
    assert(exp2(V(1)) == 2);
    assert(exp2(V(1.)) == 2);
    assert(exp2(V(1.f)) == 2);
}

__device__ void test_expm1()
{
    static_assert((std::is_same<decltype(expm1((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(expm1((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(expm1f(0)), float>::value), "");
    static_assert((std::is_same<decltype(expm1(Ambiguous())), Ambiguous>::value), "");
    assert(expm1(V(0)) == 0);
    assert(expm1(V(0.)) == 0);
    assert(expm1(V(0.f)) == 0);
}

__device__ void test_fdim()
{
    static_assert((std::is_same<decltype(fdim((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(fdim((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdimf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(fdim((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fdim(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fdim(V(1), 0) == 1);
    assert(std::fdim(V(1.), 0) == 1);
    assert(std::fdim(V(1.f), 0) == 1);

    assert(std::fdim(V(1), 0.) == 1);
    assert(std::fdim(V(1.), 0.) == 1);
    assert(std::fdim(V(1.f), 0.) == 1);

    assert(std::fdim(V(1), 0.f) == 1);
    assert(std::fdim(V(1.), 0.f) == 1);
    assert(std::fdim(V(1.f), 0.f) == 1);
}

__device__ void test_fma()
{
    static_assert((std::is_same<decltype(fma((bool)0, (float)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((char)0, (float)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((unsigned)0, (float)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((float)0, (int)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((float)0, (long)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((float)0, (float)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((float)0, (float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((float)0, (float)0, (float)0)), float>::value), "");

    static_assert((std::is_same<decltype(fma((bool)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((char)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((unsigned)0, (double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((double)0, (int)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((double)0, (long)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((double)0, (double)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((double)0, (double)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fma((double)0, (double)0,  (double)0)), double>::value), "");


    static_assert((std::is_same<decltype(fmaf(0,0,0)), float>::value), "");
    static_assert((std::is_same<decltype(fma(Ambiguous(), Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fma(V(1), 1, 1) == 2);
    assert(std::fma(V(1.), 1, 1) == 2);
    assert(std::fma(V(1.f), 1, 1) == 2);
    assert(std::fma(V(1), 1., 1) == 2);
    assert(std::fma(V(1.), 1., 1) == 2);
    assert(std::fma(V(1.f), 1., 1) == 2);
    assert(std::fma(V(1), 1.f, 1) == 2);
    assert(std::fma(V(1.), 1.f, 1) == 2);
    assert(std::fma(V(1.f), 1.f, 1) == 2);

    assert(std::fma(V(1), 1, 1.) == 2);
    assert(std::fma(V(1.), 1, 1.) == 2);
    assert(std::fma(V(1.f), 1, 1.) == 2);
    assert(std::fma(V(1), 1., 1.) == 2);
    assert(std::fma(V(1.), 1., 1.) == 2);
    assert(std::fma(V(1.f), 1., 1.) == 2);
    assert(std::fma(V(1), 1.f, 1.) == 2);
    assert(std::fma(V(1.), 1.f, 1.) == 2);
    assert(std::fma(V(1.f), 1.f, 1.) == 2);

    assert(std::fma(V(1), 1, 1.f) == 2);
    assert(std::fma(V(1.), 1, 1.f) == 2);
    assert(std::fma(V(1.f), 1, 1.f) == 2);
    assert(std::fma(V(1), 1., 1.f) == 2);
    assert(std::fma(V(1.), 1., 1.f) == 2);
    assert(std::fma(V(1.f), 1., 1.f) == 2);
    assert(std::fma(V(1), 1.f, 1.f) == 2);
    assert(std::fma(V(1.), 1.f, 1.f) == 2);
    assert(std::fma(V(1.f), 1.f, 1.f) == 2);
}

__device__ void test_fmax()
{
    static_assert((std::is_same<decltype(fmax((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(fmax((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmaxf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(fmax((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmax(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fmax(V(1), 0) == 1);
    assert(std::fmax(V(1.), 0) == 1);
    assert(std::fmax(V(1.f), 0) == 1);

    assert(std::fmax(V(1), 0.) == 1);
    assert(std::fmax(V(1.), 0.) == 1);
    assert(std::fmax(V(1.f), 0.) == 1);

    assert(std::fmax(V(1), 0.f) == 1);
    assert(std::fmax(V(1.), 0.f) == 1);
    assert(std::fmax(V(1.f), 0.f) == 1);
}

__device__ void test_fmin()
{
    static_assert((std::is_same<decltype(fmin((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(fmin((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(fminf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(fmin((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(fmin(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::fmin(V(1), 0) == 0);
    assert(std::fmin(V(1.), 0) == 0);
    assert(std::fmin(V(1.f), 0) == 0);

    assert(std::fmin(V(1), 0.) == 0);
    assert(std::fmin(V(1.), 0.) == 0);
    assert(std::fmin(V(1.f), 0.) == 0);

    assert(std::fmin(V(1), 0.f) == 0);
    assert(std::fmin(V(1.), 0.f) == 0);
    assert(std::fmin(V(1.f), 0.f) == 0);
}

__device__ void test_hypot()
{
    static_assert((std::is_same<decltype(hypot((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(hypot((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypotf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(hypot((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(hypot(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::hypot(V(3), 4) == 5);
    assert(std::hypot(V(3), 4.) == 5);
    assert(std::hypot(V(3), 4.f) == 5);

    assert(std::hypot(V(3.), 4) == 5);
    assert(std::hypot(V(3.), 4.) == 5);
    assert(std::hypot(V(3.), 4.f) == 5);

    assert(std::hypot(V(3.f), 4) == 5);
    assert(std::hypot(V(3.f), 4.) == 5);
    assert(std::hypot(V(3.f), 4.f) == 5);
}

__device__ void test_ilogb()
{
    static_assert((std::is_same<decltype(ilogb((float)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((bool)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((unsigned short)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((int)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((unsigned int)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((long)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((unsigned long)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((long long)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((unsigned long long)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb((double)0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogbf(0)), int>::value), "");
    static_assert((std::is_same<decltype(ilogb(Ambiguous())), Ambiguous>::value), "");
    assert(ilogb(V(1)) == 0);
    assert(ilogb(V(1.)) == 0);
    assert(ilogb(V(1.f)) == 0);
}

__device__ void test_lgamma()
{
    static_assert((std::is_same<decltype(lgamma((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(lgamma((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgamma((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(lgammaf(0)), float>::value), "");
    static_assert((std::is_same<decltype(lgamma(Ambiguous())), Ambiguous>::value), "");
    assert(lgamma(V(1)) == 0);
    assert(lgamma(V(1.)) == 0);
    assert(lgamma(V(1.f)) == 0);
}

__device__ void test_llrint()
{
    static_assert((std::is_same<decltype(llrint((float)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((bool)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((unsigned short)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((unsigned int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((unsigned long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((unsigned long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint((double)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrintf(0)), long long>::value), "");
    static_assert((std::is_same<decltype(llrint(Ambiguous())), Ambiguous>::value), "");
    assert(llrint(V(1)) == 1LL);
    assert(llrint(V(1.)) == 1LL);
#if CUDA_VERSION > 7050
    assert(llrint(V(1.f)) == 1LL);
#endif
}

__device__ void test_llround()
{
    static_assert((std::is_same<decltype(llround((float)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((bool)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((unsigned short)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((unsigned int)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((unsigned long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((unsigned long long)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround((double)0)), long long>::value), "");
    static_assert((std::is_same<decltype(llroundf(0)), long long>::value), "");
    static_assert((std::is_same<decltype(llround(Ambiguous())), Ambiguous>::value), "");
    assert(llround(V(1)) == 1LL);
    assert(llround(V(1.)) == 1LL);
    assert(llround(V(1.f)) == 1LL);
}

__device__ void test_log1p()
{
    static_assert((std::is_same<decltype(log1p((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(log1p((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1p((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(log1pf(0)), float>::value), "");
    static_assert((std::is_same<decltype(log1p(Ambiguous())), Ambiguous>::value), "");
    assert(log1p(V(0)) == 0);
    assert(log1p(V(0.)) == 0);
    assert(log1p(V(0.f)) == 0);
}

__device__ void test_log2()
{
    static_assert((std::is_same<decltype(log2((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(log2((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(log2f(0)), float>::value), "");
    static_assert((std::is_same<decltype(log2(Ambiguous())), Ambiguous>::value), "");
    assert(log2(V(1)) == 0);
    assert(log2(V(1.)) == 0);
    assert(log2(V(1.f)) == 0);
}

__device__ void test_logb()
{
    static_assert((std::is_same<decltype(logb((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(logb((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(logb((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(logbf(0)), float>::value), "");
    static_assert((std::is_same<decltype(logb(Ambiguous())), Ambiguous>::value), "");
    assert(logb(V(1)) == 0);
    assert(logb(V(1.)) == 0);
    assert(logb(V(1.f)) == 0);
}

__device__ void test_lrint()
{
    static_assert((std::is_same<decltype(lrint((float)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((bool)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((unsigned short)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((int)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((unsigned int)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((unsigned long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((unsigned long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint((double)0)), long>::value), "");
    static_assert((std::is_same<decltype(lrintf(0)), long>::value), "");
    static_assert((std::is_same<decltype(lrint(Ambiguous())), Ambiguous>::value), "");
    assert(lrint(V(1)) == 1L);
    assert(lrint(V(1.)) == 1L);
#if CUDA_VERSION > 7050
    assert(lrint(V(1.f)) == 1L);
#endif
}

__device__ void test_lround()
{
    static_assert((std::is_same<decltype(lround((float)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((bool)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((unsigned short)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((int)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((unsigned int)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((unsigned long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((unsigned long long)0)), long>::value), "");
    static_assert((std::is_same<decltype(lround((double)0)), long>::value), "");
    static_assert((std::is_same<decltype(lroundf(0)), long>::value), "");
    static_assert((std::is_same<decltype(lround(Ambiguous())), Ambiguous>::value), "");
    assert(lround(V(1)) == 1L);
    assert(lround(V(1.)) == 1L);
    assert(lround(V(1.f)) == 1L);
}

__device__ void test_nan()
{
    static_assert((std::is_same<decltype(nan("")), double>::value), "");
    static_assert((std::is_same<decltype(nanf("")), float>::value), "");
}

__device__ void test_nearbyint()
{
    static_assert((std::is_same<decltype(nearbyint((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(nearbyint((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyint((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(nearbyintf(0)), float>::value), "");
    static_assert((std::is_same<decltype(nearbyint(Ambiguous())), Ambiguous>::value), "");
    assert(nearbyint(V(1)) == 1);
    assert(nearbyint(V(1.)) == 1);
    assert(nearbyint(V(1.f)) == 1);
    // There are more checks in test_rint(). rint and nearbyint behave the same
    // way on the GPU, so we only test them in one place.
}

__device__ void test_nextafter()
{
    static_assert((std::is_same<decltype(nextafter((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(nextafter((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafterf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(nextafter((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(nextafter(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    //assert(nextafter(0,1) == hexfloat<double>(0x1, 0, -1074));

    // Invoke all our overloads.  Even though we don't check the exact result
    // (this is pretty annoying to do for this function), we make sure to *use*
    // the results so that these function calls can't be DCE'ed.
    assert(nextafter(V(0), 1) != 0);
    assert(nextafter(V(0), 1.) != 0);
    assert(nextafter(V(0), 1.f) != 0);

    assert(nextafter(V(0.), 1) != 0);
    assert(nextafter(V(0.), 1.) != 0);
    assert(nextafter(V(0.), 1.f) != 0);

    assert(nextafter(V(0.f), 1) != 0);
    assert(nextafter(V(0.f), 1.) != 0);
    assert(nextafter(V(0.f), 1.f) != 0);
}

__device__ void test_remainder()
{
    static_assert((std::is_same<decltype(remainder((float)0, (float)0)), float>::value), "");
    static_assert((std::is_same<decltype(remainder((bool)0, (float)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((unsigned short)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((float)0, (unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((int)0, (long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((int)0, (unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((double)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder((float)0, (double)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainderf(0,0)), float>::value), "");
    static_assert((std::is_same<decltype(remainder((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(remainder(Ambiguous(), Ambiguous())), Ambiguous>::value), "");

    assert(std::remainder(V(1.5), 1) == -.5);
    assert(std::remainder(V(1.5), 1.) == -.5);
    assert(std::remainder(V(1.5), 1.f) == -.5);

    assert(std::remainder(V(1.5f), 1) == -.5);
    assert(std::remainder(V(1.5f), 1.) == -.5);
    assert(std::remainder(V(1.5f), 1.f) == -.5);

    assert(std::remainder(V(2), 1) == 0);
    assert(std::remainder(V(2), 1.) == 0);
    assert(std::remainder(V(2), 1.f) == 0);
}

__device__ void test_remquo()
{
    int ip;
    static_assert((std::is_same<decltype(remquo((float)0, (float)0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(remquo((bool)0, (float)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((unsigned short)0, (double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((float)0, (unsigned int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((double)0, (long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((int)0, (long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((int)0, (unsigned long long)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((double)0, (double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo((float)0, (double)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquof(0,0, &ip)), float>::value), "");
    static_assert((std::is_same<decltype(remquo((int)0, (int)0, &ip)), double>::value), "");
    static_assert((std::is_same<decltype(remquo(Ambiguous(), Ambiguous(), &ip)), Ambiguous>::value), "");

    assert(std::remquo(V(1), 1, &ip) == 0);
    assert(std::remquo(V(1), 1., &ip) == 0);
    assert(std::remquo(V(1), 1.f, &ip) == 0);

    assert(std::remquo(V(0.5), 1, &ip) == 0.5);
    assert(std::remquo(V(0.5), 1., &ip) == 0.5);
    assert(std::remquo(V(0.5), 1.f, &ip) == 0.5);

    assert(std::remquo(V(0.5f), 1, &ip) == 0.5);
    assert(std::remquo(V(0.5f), 1., &ip) == 0.5);
    assert(std::remquo(V(0.5f), 1.f, &ip) == 0.5);
}

__device__ void test_rint_nearbyint()
{
    static_assert((std::is_same<decltype(rint((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(rint((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(rint((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(rintf(0)), float>::value), "");
    static_assert((std::is_same<decltype(rint(Ambiguous())), Ambiguous>::value), "");
    // Verify that rint/nearbyint produce identical correct results
    auto check = [](double input, double fpresult) {
      // FP rint()/nearbyint must match the expected result.
      assert(rint(V(float(input))) == float(fpresult));
      assert(nearbyint(V(float(input))) == float(fpresult));
      assert(rint(V(input)) == fpresult);
      assert(nearbyint(V(input)) == fpresult);
      // for integral types, std::rint(input) == std::rint(double(input))
      int iinput = input;
      assert(std::rint(V(iinput)) == std::rint(double(V(iinput))));
      assert(std::nearbyint(V(iinput)) == std::nearbyint(double(V(iinput))));
    };
    // Whole values round to themselves and do not change sign.
    check(0.0, 0.0);
    check(-0.0, -0.0);
    check(1.0, 1.0);
    check(-1.0, -1.0);
    // Half-way values round towards nearest even number.
    check(2.5, 2.0);
    check(-2.5, -2.0);
    check(3.5, 4.0);
    check(-3.5, -4.0);
    // Everything else is rounded towards nearest integer.
    check(2.1, 2.0);
    check(-2.1, -2.0);
    check(2.7, 3.0);
    check(-2.7, -3.0);
    check(3.9, 4.0);
    check(-3.9, -4.0);
}

__device__ void test_round()
{
    static_assert((std::is_same<decltype(round((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(round((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(round((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(roundf(0)), float>::value), "");
    static_assert((std::is_same<decltype(round(Ambiguous())), Ambiguous>::value), "");
    assert(round(V(1)) == 1);
    assert(round(V(1.)) == 1);
    assert(round(V(1.f)) == 1);
}

__device__ void test_scalbln()
{
    static_assert((std::is_same<decltype(scalbln((float)0, (long)0)), float>::value), "");
    static_assert((std::is_same<decltype(scalbln((bool)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((unsigned short)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((int)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((unsigned int)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((unsigned long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((long long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((unsigned long long)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbln((double)0, (long)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalblnf(0, (long)0)), float>::value), "");
    static_assert((std::is_same<decltype(scalbln(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::scalbln(V(1), 1) == 2);
    assert(std::scalbln(V(1), 1.) == 2);
    assert(std::scalbln(V(1), 1.f) == 2);

    assert(std::scalbln(V(1.), 1) == 2);
    assert(std::scalbln(V(1.), 1.) == 2);
    assert(std::scalbln(V(1.), 1.f) == 2);

    assert(std::scalbln(V(1.f), 1) == 2);
    assert(std::scalbln(V(1.f), 1.) == 2);
    assert(std::scalbln(V(1.f), 1.f) == 2);
}

__device__ void test_scalbn()
{
    static_assert((std::is_same<decltype(scalbn((float)0, (int)0)), float>::value), "");
    static_assert((std::is_same<decltype(scalbn((bool)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((unsigned short)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((unsigned int)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((unsigned long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((long long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((unsigned long long)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbn((double)0, (int)0)), double>::value), "");
    static_assert((std::is_same<decltype(scalbnf(0, (int)0)), float>::value), "");
    static_assert((std::is_same<decltype(scalbn(Ambiguous(), Ambiguous())), Ambiguous>::value), "");
    assert(std::scalbn(V(1), 1) == 2);
    assert(std::scalbn(V(1), 1.) == 2);
    assert(std::scalbn(V(1), 1.f) == 2);

    assert(std::scalbn(V(1.), 1) == 2);
    assert(std::scalbn(V(1.), 1.) == 2);
    assert(std::scalbn(V(1.), 1.f) == 2);

    assert(std::scalbn(V(1.f), 1) == 2);
    assert(std::scalbn(V(1.f), 1.) == 2);
    assert(std::scalbn(V(1.f), 1.f) == 2);
}

__device__ void test_tgamma()
{
    static_assert((std::is_same<decltype(tgamma((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(tgamma((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgamma((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(tgammaf(0)), float>::value), "");
    static_assert((std::is_same<decltype(tgamma(Ambiguous())), Ambiguous>::value), "");
    assert(tgamma(V(1)) == 1);
    assert(tgamma(V(1.)) == 1);
    assert(tgamma(V(1.f)) == 1);
}

__device__ void test_trunc()
{
    static_assert((std::is_same<decltype(trunc((float)0)), float>::value), "");
    static_assert((std::is_same<decltype(trunc((bool)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((unsigned short)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((int)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((unsigned int)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((long)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((unsigned long)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((unsigned long long)0)), double>::value), "");
    static_assert((std::is_same<decltype(trunc((double)0)), double>::value), "");
    static_assert((std::is_same<decltype(truncf(0)), float>::value), "");
    static_assert((std::is_same<decltype(trunc(Ambiguous())), Ambiguous>::value), "");
    assert(trunc(V(1)) == 1);
    assert(trunc(V(1.)) == 1);
    assert(trunc(V(1.f)) == 1);
}

__global__ void tests()
{
    test_abs();
    test_acos();
    test_asin();
    test_atan();
    test_atan2();
    test_ceil();
    test_cos();
    test_cosh();
    test_exp();
    test_fabs();
    test_floor();
    test_fmod();
    test_frexp();
    test_ldexp();
    test_log();
    test_log10();
    test_modf();
    test_pow();
    test_sin();
    test_sinh();
    test_sqrt();
    test_tan();
    test_tanh();
    test_signbit();
    test_fpclassify();
    test_isfinite();
    test_isnormal();
    test_isgreater();
    test_isgreaterequal();
    test_isinf();
    test_isless();
    test_islessequal();
    test_islessgreater();
    test_isnan();
    test_isunordered();
    test_acosh();
    test_asinh();
    test_atanh();
    test_cbrt();
    test_copysign();
    test_erf();
    test_erfc();
    test_exp2();
    test_expm1();
    test_fdim();
    test_fma();
    test_fmax();
    test_fmin();
    test_hypot();
    test_ilogb();
    test_lgamma();
    test_llrint();
    test_llround();
    test_log1p();
    test_log2();
    test_logb();
    test_lrint();
    test_lround();
    test_nan();
    test_nearbyint();
    test_nextafter();
    test_remainder();
    test_remquo();
    test_rint_nearbyint();
    test_round();
    test_scalbln();
    test_scalbn();
    test_tgamma();
    test_trunc();
}

int main() {
    tests<<<1,1>>>();
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      printf("CUDA error %d\n", (int)err);
      return 1;
    }
    printf("Success!\n");
    return 0;
}


#else

#include <stdio.h>

// No C++11; test is a nop.
int main() {
  printf("Success!\n");
  return 0;
}

#endif // __cplusplus < 201103L
